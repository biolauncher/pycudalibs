#include "hip/hip_runtime.h"
/*
* Null kernel to get library started
*/
#include <hip/hip_runtime_api.h>
#include <cudaml.h>

__global__ void null_kernel(void) { }

 
hipError_t null(void) {
  null_kernel<<<1,1>>> ();
  return hipGetLastError();
}


